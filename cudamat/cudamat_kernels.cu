#include "hip/hip_runtime.h"
#include "cudamat_kernels.cuh"
#include "float.h"

/* ------------------------- Random number generation ------------------------- */

__global__ void cudamat_kSeedRandom(unsigned int* rndMults, unsigned long long* rndWords, unsigned int seed) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // The initial x is the seed and the initial carry is 1
    unsigned long long rndWord = ((unsigned long long)seed << 32) + 1;
    const unsigned int rndMult = rndMults[idx];
    /*
     * Run the chain for a few steps so that all the streams have a chance
     * to differentiate. They start out generating similar random numbers
     * because all the multipliers are similar.
     */
    for(unsigned int i = 0; i < NUM_RND_BURNIN; i++) {
        rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
    }
    rndWords[idx] = rndWord;
}

__global__ void cudamat_kRandomUniform(unsigned int* rndMults, unsigned long long* rndWords, float* gData, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long rndWord = rndWords[idx];
    const unsigned int rndMult = rndMults[idx];

    for(unsigned int i = idx; i < numElements; i += NUM_RND_STREAMS) {
        rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
        gData[i] = (__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
    }
    rndWords[idx] = rndWord;
}

__global__ void cudamat_kRandomGaussian(unsigned int* rndMults, unsigned long long* rndWords, float* gData, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long rndWord = rndWords[idx];
    const unsigned int rndMult = rndMults[idx];

    float rnd1, rnd2, R, T;
    for(unsigned int i = idx; i < numElements; i += 2*NUM_RND_STREAMS) {
        rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
        rnd1 = (__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
        rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
        rnd2 = (__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
        T = 2 * PI * rnd2;
        R = sqrtf(-2 * __logf(rnd1));
        gData[i] = R * __cosf(T);
        if (i + NUM_RND_STREAMS < numElements)
            gData[i + NUM_RND_STREAMS] = R * __sinf(T);
    }
    rndWords[idx] = rndWord;
}

/* ------------------------- Data copying ------------------------- */

/*
Copy row slice from source to target. There is a block for every 32x32 chunk being copied.
*/
__global__ void cudamat_kGetRowSlice(float* source, float* target, int start, int end, int width, int height) {
    const int row = start + blockIdx.x * 32 + threadIdx.x;
    const int start_col = blockIdx.y * 32;

    const int end_col = (start_col + 32 < width) ? start_col + 32: width;

    const int target_height = end - start;

    if (row < end) {
        for (int cur_col = start_col; cur_col < end_col; cur_col++)
            target[cur_col * target_height + row - start] = source[cur_col * height + row];
    }
}

__global__ void cudamat_kSetRowSlice(float* source, float* target, int start, int end, int width, int height) {
    const int row = start + blockIdx.x * 32 + threadIdx.x;
    const int start_col = blockIdx.y * 32;

    const int end_col = (start_col + 32 < width) ? start_col + 32: width;

    const int source_height = end - start;

    if (row < end) {
        for (int cur_col = start_col; cur_col < end_col; cur_col++)
            target[cur_col * height + row] = source[cur_col * source_height + row - start];
            //source[cur_col * height + row - start] = target[cur_col * target_height + row];
    }
}

__global__ void cudamat_kTranspose(float *odata, float *idata, int width, int height) {
    __shared__ float block[COPY_BLOCK_SIZE][COPY_BLOCK_SIZE+1];

    // read the matrix tile into shared memory
    unsigned int xIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.x;
    unsigned int yIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.y;

    if((xIndex < width) && (yIndex < height)) {
        unsigned int index_in = yIndex * width + xIndex;

        block[threadIdx.y][threadIdx.x] = idata[index_in];
    }

    __syncthreads();

    // write the transposed matrix tile to global memory
    xIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.x;
    yIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.y;

    if((xIndex < height) && (yIndex < width)) {
        unsigned int index_out = yIndex * height + xIndex;

        odata[index_out] = block[threadIdx.x][threadIdx.y];
    }
}

/* ------------------------- Mathematical operations ------------------------- */

__global__ void cudamat_kLessThan(float* mat1, float* mat2, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = mat1[i] < mat2[i];
    }
}

__global__ void cudamat_kLessThanScalar(float* mat, float val, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = mat[i] < val;
    }
}

__global__ void cudamat_kGreaterThan(float* mat1, float* mat2, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = mat1[i] > mat2[i];
    }
}

__global__ void cudamat_kGreaterThanScalar(float* mat, float val, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = mat[i] > val;
    }
}

__global__ void cudamat_kMaxColumnwise(float* mat, float* target, unsigned int width, unsigned int height) {
    __shared__ float max_vals[32];
    float cur_max = -FLT_MAX;
    float val = 0;
 
    for (unsigned int i = threadIdx.x; i < height; i += 32) {
        val = mat[blockIdx.x * height + i];

        if (val > cur_max)
            cur_max = val;
    }

    max_vals[threadIdx.x] = cur_max;

    __syncthreads();

    if (threadIdx.x == 0) {
        cur_max = -FLT_MAX;

        for (unsigned int i = 0; i < 32; i++)
            if (max_vals[i] > cur_max)
                cur_max = max_vals[i];

        target[blockIdx.x] = cur_max;
    }
}


__global__ void cudamat_kRowArgmax(float* mat, 
				   float* target_max, 
				   float* target_arg_max,
				   unsigned int width, unsigned int height) {
    __shared__ float max_vals[32];
    __shared__ float max_inds[32];
    float cur_max = -FLT_MAX;
    int ind_max = 0;
    float val = 0;
 
    for (unsigned int i = threadIdx.x; i < height; i += 32) {
        val = mat[blockIdx.x * height + i];

        if (val > cur_max){
            cur_max = val;
	    ind_max = i;
	}
    }

    max_vals[threadIdx.x] = cur_max;
    max_inds[threadIdx.x] = ind_max;

    __syncthreads();

    if (threadIdx.x == 0) {
        cur_max = -FLT_MAX;

        for (unsigned int i = 0; i < 32; i++)
	  if (max_vals[i] > cur_max){
                cur_max = max_vals[i];
		ind_max = max_inds[i];
	  }

        target_max[blockIdx.x] = cur_max;
	target_arg_max[blockIdx.x * height + ind_max] = 1; // that's the way to go. Yes. Now we have the argmax.
    }
}


__global__ void cudamat_kSign(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = mat[i] ? copysignf(1., mat[i]) : 0.;
    }
}

__global__ void cudamat_kApplySigmoid(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = 1 / (1 + __expf(-mat[i]));
    }
}


__global__ void cudamat_kApplyTanh(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    float mat_i, exp2x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        mat_i = mat[i];
        exp2x = __expf(2 * mat_i);
        target[i] = 1 - 2 / (exp2x + 1);
    }
}

__global__ void cudamat_kApplyAbs(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    
    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = mat[i] * ((mat[i] > 0) - (mat[i] < 0));
    }
}

__global__ void cudamat_kApplyLog1PlusExp(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    float mat_i;

    for (unsigned int i = idx; i < len; i += numThreads) {
        mat_i = mat[i];
        if (mat_i > 0)
            target[i] = (__logf(1 + __expf(-mat_i)) + mat_i);
        else
            target[i] = __logf(1 + __expf(mat_i));
    }
}

__global__ void cudamat_kLog(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = __logf(mat[i]);
    }
}

__global__ void cudamat_kExp(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = __expf(mat[i]);
    }
}

__global__ void cudamat_kSqrt(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = sqrt(mat[i]);
    }
}

__global__ void cudamat_kPow(float* mat, float pow, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = powf(mat[i], pow);
    }
}

__global__ void cudamat_kPowMatrix(float* mat, float* pow, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = powf(mat[i], pow[i]);
    }
}

__global__ void cudamat_kReciprocal(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads)
        target[i] = 1. / mat[i];
}

__global__ void cudamat_kAddColVector(float* mat, float* vec, float* tgtMat, unsigned int width,
                              unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < width * height; i += numThreads) {
        tgtMat[i] = mat[i] + vec[i % height];
    }
}

__global__ void cudamat_kAddRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < width * height; i += numThreads) {
        tgtMat[i] = mat[i] + vec[i / height];
    }
}

__global__ void cudamat_kAddColMult(float* mat, float* vec, float* tgtMat, float mult,
                            unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < width * height; i += numThreads) {
        tgtMat[i] = mat[i] + mult * vec[i % height];
    }
}

__global__ void cudamat_kMultByColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < width * height; i += numThreads) {
        tgtMat[i] = mat[i] * vec[i % height];
    }
}

__global__ void cudamat_kMultByRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < width * height; i += numThreads) {
        tgtMat[i] = mat[i] * vec[i / height];
    }
}

__global__ void cudamat_kAdd(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < numEls; i += numThreads) {
        dest[i] = a[i] + b[i];
    }
}

__global__ void cudamat_kSubtract(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < numEls; i += numThreads) {
        dest[i] = a[i] - b[i];
    }
}

__global__ void cudamat_kDivide(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < numEls; i += numThreads) {
        dest[i] = a[i] / b[i];
    }
}

__global__ void cudamat_kMult(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < numEls; i += numThreads) {
        dest[i] = a[i] * b[i];
    }
}

__global__ void cudamat_kMultScalar(float* mat, float alpha, float* dest, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        dest[i] = alpha * mat[i];
    }
}

__global__ void cudamat_kAssignScalar(float* dest, float alpha, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        dest[i] = alpha;
    }
}

__global__ void cudamat_kDivideScalar(float* mat, float alpha, float* dest, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        dest[i] = mat[i] / alpha;
    }
}

__global__ void cudamat_kAddScalar(float* a, float alpha, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < numEls; i += numThreads) {
        dest[i] = a[i] + alpha;
    }
}

// __global__ void cudamat_kSelectRows(float* source, float* target, float* indices, int nRowIs, int nCols, int nSourceRows){
//     __shared__ int sourceRowIndices[32];
//     const int startTargetRowI = blockIdx.x * 32;
//     const int tid = threadIdx.x;
//     const int localNRowIs = min(32, nRowIs-startTargetRowI);

//     // cooperatively load 32 row indices
//     if (tid < localNRowIs){
//         sourceRowIndices[tid] = int(indices[startTargetRowI + tid]);
//         if (sourceRowIndices[tid]<0)
//             sourceRowIndices[tid] += nSourceRows;
//         if (sourceRowIndices[tid]<0 || sourceRowIndices[tid]>=nSourceRows)
//             sourceRowIndices[tid] = -1;
//     }
//     __syncthreads();

//     // copy 32 rows
//     for (int i=0; i<localNRowIs; i++){
//         const int targetRowI = startTargetRowI + i, sourceRowI = sourceRowIndices[i];
//         for (int colI=tid; colI<nCols; colI+=32)
//             target[targetRowI * nCols + colI] = sourceRowI==-1 ? (1.0/0.0 -1.0/0.0) : source[sourceRowI * nCols + colI];
//     }
// }

__global__ void cudamat_kSetSelectedRows(float* target, float* source, float* indices, int nRowIs, int nCols, int nTargetRows){
    __shared__ int targetRowIndices[32];
    const int startSourceRowI = blockIdx.x * 32;
    const int tid = threadIdx.x;
    const int localNRowIs = min(32, nRowIs-startSourceRowI);

    // cooperatively load 32 row indices
    if (tid < localNRowIs){
        targetRowIndices[tid] = int(indices[startSourceRowI + tid]);
        if (targetRowIndices[tid]<0)
            targetRowIndices[tid] += nTargetRows;
        if (targetRowIndices[tid]<0 || targetRowIndices[tid]>=nTargetRows)
            targetRowIndices[tid] = -1;
    }
    __syncthreads();

    // copy 32 rows
    for (int i=0; i<localNRowIs; i++){
        const int sourceRowI = startSourceRowI + i, targetRowI = targetRowIndices[i];
        for (int colI=tid; colI<nCols; colI+=32)
            target[targetRowI * nCols + colI] = targetRowI==-1 ? (1.0/0.0 -1.0/0.0) : source[sourceRowI * nCols + colI];
    }
}


__global__ void cudamat_kSelectRows(float* source, float* target, float* indices, int nRowIs, int nCols, int nSourceRows){
    __shared__ int sourceRowIndices[32];
    const int tid = threadIdx.x, bid = blockIdx.x*gridDim.y+blockIdx.y;
    const int startTargetRowI = bid * 32;
    if (startTargetRowI>=nRowIs) return; // unneeded block
    const int localNRowIs = min(32, nRowIs-startTargetRowI);

    // cooperatively load 32 row indices
    if (tid < localNRowIs){
        sourceRowIndices[tid] = int(indices[startTargetRowI + tid]);
        if (sourceRowIndices[tid]<0)
            sourceRowIndices[tid] += nSourceRows;
        if (sourceRowIndices[tid]<0 || sourceRowIndices[tid]>=nSourceRows)
            sourceRowIndices[tid] = -1;
    }
    __syncthreads();

    // copy 32 rows
    for (int i=0; i<localNRowIs; i++){
        const int targetRowI = startTargetRowI + i, sourceRowI = sourceRowIndices[i];
        for (int colI=tid; colI<nCols; colI+=32)
            target[targetRowI * nCols + colI] = sourceRowI==-1 ? (1.0/0.0 -1.0/0.0) : source[sourceRowI * nCols + colI];
    }
}

__global__ void cudamat_kClfPcOuterProduct(int maxNIndexPairs, float* GindexPairs, float* nIndexPairss, float* A, float* B, float* ret, int nCols, int nBlocks){
  /*
    a block per row. a thread per ret cell (except: at least 32 threads).
    dynamically allocated shared mem: two words per index pair.
  */
  const int tid = threadIdx.x, bid = blockIdx.x*gridDim.y+blockIdx.y;
  if (bid>=nBlocks) return;

  const int nIndexPairs = nIndexPairss[bid];
  extern __shared__ int indexPairs[];

  // cooperatively load index pairs
  GindexPairs += bid * maxNIndexPairs*2;
  const int nValuesToCopy = nIndexPairs * 2;
  if (tid<32) for (int i=tid; i<nValuesToCopy; i+=32) indexPairs[i] = GindexPairs[i];
  __syncthreads();

  // from now on, everything is thread local
  if (tid<nCols){
    float cum=0;
    for (int iPairI=0; iPairI<nIndexPairs; iPairI++){
      const int aI = indexPairs[iPairI*2], bI = indexPairs[iPairI*2+1];
      cum += A[nCols*aI + tid] * B[nCols*bI + tid];
    }
    ret[nCols*bid + tid] = cum;
  }
}

__global__ void cudamat_kClfVsProduct(int nComponents, int vectorLength, int nothingIndex_scalars, float* inVectors, float* outVectors, float* globalScalars, float* inIndices, int nBlocks){
  __shared__ int vectorIndices[32];
  __shared__ float localScalars[32];
  
  const int tid = threadIdx.x, bid = blockIdx.x*gridDim.y+blockIdx.y;
  if (bid>=nBlocks) return;
  inIndices += bid * 2 * nComponents;
  
  float cum=0;
  for (int componentI=0; componentI<nComponents; componentI++){
    if ((componentI & 31) == 0){ // download the next 32 scalars & indices
      if (tid<32 && componentI+tid<nComponents){
	vectorIndices[tid] = inIndices[componentI+tid];
	const int scalarIndex = inIndices[componentI+tid + nComponents];
	localScalars[tid] = (scalarIndex==nothingIndex_scalars) ? 0 : globalScalars[scalarIndex];
      }
    }
    __syncthreads(); // this should of course be in the conditional that just closed, but putting it there seems to make it not work.
    if ((tid < vectorLength) && (localScalars[componentI & 31] != 0))
      cum += localScalars[componentI & 31] * inVectors[vectorIndices[componentI & 31] * vectorLength + tid];
  }
  if (tid < vectorLength)
    outVectors[bid * vectorLength + tid] = cum;
}
    

