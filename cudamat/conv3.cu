#include "hip/hip_runtime.h"
/*
 * conv3.cu
 *
 *  Created on: Nov 15, 2009
 *      Author: Alex Krizhevsky (akrizhevsky@gmail.com)
 */
#include <math.h>
#include <nvmatrix.cuh>
#include "conv3.cuh"

void _convolve3_bw(float* images, float* filters, float* targets, int numImgsPerGroup,
                  int numFiltersPerGroup, int numGroups, int imgSize, int filterSize, int stride, ORDER imgOrder) {
    assert(stride == 1 || stride == 3);
    int numOutputsX = imgSize - filterSize + 1;
//    int numOutputs = numOutputsX*numOutputsX;
    bool checkOutputBounds = numOutputsX % 16 != 0;
    if(filterSize > 37) {
        int numPartsX = DIVUP(numOutputsX, 16);
        int numParts = numPartsX*numPartsX;
        int blocksY = numParts, blocksX = numImgsPerGroup * numGroups;
        dim3 grid(blocksX, blocksY);
        dim3 threads(16, 16);
        bool checkFilterBounds = filterSize % 16 != 0;
//        printf("check filter bounds: %d, check output bounds: %d, stride: %d\n", checkFilterBounds, checkOutputBounds, stride);
        if (imgOrder == GROUP_FILTER_IMAGE) {
            if(checkFilterBounds) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_nofit_16x16<true, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, filterSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_nofit_16x16<true, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, filterSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_nofit_16x16<false, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, filterSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_nofit_16x16<false, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, filterSize, numFiltersPerGroup, numGroups);
                    }
                }
            } else {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_nofit_16x16<true, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, filterSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_nofit_16x16<true, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, filterSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_nofit_16x16<false, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, filterSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_nofit_16x16<false, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, filterSize, numFiltersPerGroup, numGroups);
                    }
                }
            }
        } else {
            if(checkFilterBounds) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_nofit_16x16<true, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, filterSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_nofit_16x16<true, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, filterSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_nofit_16x16<false, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, filterSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_nofit_16x16<false, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, filterSize, numFiltersPerGroup, numGroups);
                    }
                }
            } else {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_nofit_16x16<true, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, filterSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_nofit_16x16<true, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, filterSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_nofit_16x16<false, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, filterSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_nofit_16x16<false, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, filterSize, numFiltersPerGroup, numGroups);
                    }
                }
            }
        }
    } else {
        int numPartsX = DIVUP(numOutputsX, 16);
        int numParts = numPartsX*numPartsX;
        int blocksY = numParts, blocksX = numImgsPerGroup * numGroups;
        dim3 grid(blocksX, blocksY);
        dim3 threads(16, 16);
//            printf("numFiltersPerGroup: %d, numImgsPerGroup: %d, numGroups: %d\n", numFiltersPerGroup, numImgsPerGroup, numGroups);
//            printf("blocksX: %d\n", blocksX);
//            printf("stride: %d\n", stride);
        /*
         * This code was auto-generated...
         */
        if(imgOrder == GROUP_FILTER_IMAGE) {
            if (filterSize == 1) {
                throw "try multByScalar";
            } else if (filterSize == 2) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<2, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<2, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<2, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<2, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 3) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<3, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<3, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<3, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<3, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 4) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<4, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<4, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<4, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<4, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 5) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<5, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<5, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<5, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<5, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 6) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<6, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<6, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<6, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<6, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 7) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<7, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<7, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<7, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<7, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 8) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<8, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<8, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<8, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<8, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 9) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<9, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<9, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<9, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<9, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 10) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<10, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<10, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<10, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<10, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 11) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<11, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<11, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<11, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<11, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 12) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<12, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<12, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<12, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<12, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 13) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<13, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<13, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<13, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<13, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 14) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<14, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<14, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<14, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<14, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 15) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<15, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<15, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<15, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<15, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 16) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<16, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<16, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<16, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<16, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 17) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<17, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<17, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<17, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<17, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 18) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<18, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<18, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<18, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<18, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 19) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<19, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<19, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<19, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<19, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 20) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<20, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<20, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<20, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<20, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 21) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<21, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<21, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<21, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<21, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 22) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<22, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<22, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<22, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<22, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 23) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<23, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<23, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<23, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<23, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 24) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<24, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<24, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<24, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<24, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 25) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<25, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<25, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<25, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<25, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 26) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<26, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<26, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<26, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<26, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 27) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<27, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<27, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<27, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<27, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 28) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<28, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<28, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<28, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<28, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 29) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<29, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<29, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<29, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<29, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 30) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<30, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<30, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<30, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<30, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 31) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<31, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<31, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<31, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<31, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 32) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<32, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<32, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<32, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<32, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 33) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<33, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<33, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<33, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<33, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 34) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<34, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<34, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<34, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<34, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 35) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<35, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<35, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<35, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<35, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 36) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<36, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<36, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<36, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<36, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 37) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<37, true, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<37, true, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<37, false, 1, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<37, false, 3, true><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }
        } else {
            if (filterSize == 1) {
                throw "try multByScalar";
            } else if (filterSize == 2) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<2, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<2, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<2, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<2, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 3) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<3, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<3, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<3, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<3, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 4) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<4, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<4, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<4, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<4, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 5) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<5, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<5, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<5, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<5, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 6) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<6, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<6, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<6, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<6, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 7) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<7, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<7, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<7, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<7, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 8) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<8, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<8, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<8, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<8, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 9) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<9, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<9, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<9, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<9, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 10) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<10, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<10, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<10, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<10, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 11) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<11, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<11, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<11, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<11, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 12) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<12, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<12, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<12, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<12, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 13) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<13, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<13, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<13, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<13, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 14) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<14, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<14, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<14, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<14, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 15) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<15, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<15, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<15, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<15, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 16) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<16, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<16, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<16, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<16, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 17) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<17, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<17, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<17, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<17, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 18) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<18, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<18, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<18, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<18, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 19) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<19, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<19, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<19, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<19, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 20) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<20, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<20, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<20, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<20, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 21) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<21, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<21, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<21, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<21, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 22) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<22, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<22, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<22, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<22, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 23) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<23, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<23, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<23, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<23, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 24) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<24, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<24, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<24, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<24, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 25) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<25, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<25, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<25, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<25, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 26) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<26, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<26, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<26, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<26, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 27) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<27, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<27, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<27, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<27, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 28) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<28, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<28, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<28, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<28, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 29) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<29, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<29, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<29, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<29, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 30) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<30, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<30, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<30, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<30, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 31) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<31, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<31, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<31, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<31, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 32) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<32, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<32, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<32, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<32, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 33) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<33, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<33, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<33, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<33, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 34) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<34, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<34, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<34, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<34, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 35) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<35, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<35, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<35, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<35, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 36) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<36, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<36, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<36, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<36, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }  else if (filterSize == 37) {
                if (checkOutputBounds) {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<37, true, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<37, true, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                } else {
                    if (stride == 1) {
                        conv3_bw_fit_16x16<37, false, 1, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    } else {
                        conv3_bw_fit_16x16<37, false, 3, false><<<grid, threads>>>(images, filters, targets, imgSize, numFiltersPerGroup, numGroups);
                    }
                }
            }
        }
    }
    cutilCheckMsg("kernel execution failed");
}

/*
 * The input matrices must have these shapes:
 *
 * GROUP_FILTER_IMAGE:
    Matrix images(numFiltersPerGroup * numGroups, numImgsPerGroup * imgPixels);
    Matrix filters(numFiltersPerGroup * numGroups, filterPixels * colorMult);
    Matrix targets(numImgsPerGroup * numGroups, numOutputs*colorMult);

   IMAGE_GROUP_FILTER:
    Matrix images(numImgsPerGroup * numGroups, numFiltersPerGroup * imgPixels);
    Matrix filters(numFiltersPerGroup * numGroups, filterPixels * colorMult);
    Matrix targets(numImgsPerGroup * numGroups, numOutputs*colorMult);
 */
void convolve3(NVMatrix* images, NVMatrix* filters, NVMatrix* targets, int numGroups, bool color, ORDER imgOrder) {
    assert(targets->getNumRows() % numGroups == 0);
    int colorMult = color ? 3 : 1;
    assert(filters->getNumCols() % colorMult == 0);
    assert(filters->getNumRows() % numGroups == 0);
    int numFiltersPerGroup = filters->getNumRows() / numGroups;
    int numImgsPerGroup = targets->getNumRows() / numGroups;

    if (imgOrder == GROUP_FILTER_IMAGE) {
        assert(images->getNumCols() % numImgsPerGroup == 0);
        assert(images->getNumRows() == filters->getNumRows());
    } else {
        assert(images->getNumRows() == numImgsPerGroup * numGroups);
        assert(images->getNumCols() % numFiltersPerGroup == 0);
    }

//    assert(filters->getNumCols() % numFilters == 0);
    int imgPixels = imgOrder == GROUP_FILTER_IMAGE ? images->getNumCols() / numImgsPerGroup
                                                   : images->getNumCols() / numFiltersPerGroup;
    int filterPixels = filters->getNumCols() / colorMult;
    assert(sqrt(double(imgPixels)) == floor(sqrt(double(imgPixels))));
    assert(sqrt(double(filterPixels)) == floor(sqrt(double(filterPixels))));
    int imgSize = int(sqrt(double(imgPixels)));
    int filterSize = int(sqrt(double(filterPixels)));

    int numOutputsX = imgSize - filterSize + 1;
    int numOutputs = numOutputsX * numOutputsX;
    assert(targets->getNumCols() == numOutputs * colorMult);
//    assert(targets->getNumElements() == numOutputs * numImgsPerGroup * numGroups * colorMult);
    assert(!images->isTrans());
    assert(!filters->isTrans());
    assert(!targets->isTrans());
    assert(imgSize > filterSize);

//    printf("computed numcases: %d, numfilters: %d, imgsize: %d, filtersize: %d\n", numCases, numFilters, imgSize, filterSize);
    _convolve3_bw(images->getDevData(), filters->getDevData(), targets->getDevData(),
                 numImgsPerGroup * colorMult, numFiltersPerGroup, numGroups, imgSize, filterSize, colorMult, imgOrder);
}
