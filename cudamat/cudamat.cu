#include "hip/hip_runtime.h"
#define FULL
/*
 * testconv.cu
 *
 *  Created on: Oct 31, 2009
 *      Author: Alex Krizhevsky (akrizhevsky@gmail.com)
 */

#ifdef FULL
#include <cutil_inline.h>
#include <assert.h>
#include <nvmatrix.cuh>
#include <matrix.h>

//#include "testconv_extras.cuh"
#include "conv.cuh"
//#include "conv2.cuh"
#include "conv_util.cuh"
#include "conv3.cuh"
#include "convCPU.h"
#include "gpu_locking.h"

static uint timer;

void init_tests(int boardNum) {
    hipSetDevice(boardNum > -1 ? boardNum : cutGetMaxGflopsDeviceId());
    cublasInit();
    NVMatrix::initDeviceProps();
    NVMatrix::initRandom(7);
    //cutilCheckError(cutCreateTimer( &timer));
}

void test_convolve(int imgSize, int filterSize, bool color) {
    printf("===============================\n");
    printf("test_convolve\n");
    printf("===============================\n");

    ORDER order = IMAGE_GROUP_FILTER;
    int numFiltersPerGroup = 64, numImgsPerGroup = 128, numGroups = 4;
    int filterPixels = filterSize * filterSize;
    int imgPixels = imgSize * imgSize;
    int numOutputsX = imgSize - filterSize + 1;
    int numOutputs = numOutputsX * numOutputsX;
//    assert(numFiltersPerGroup % 8 == 0);
    printf("Groups: %d\n", numGroups);
    printf("Images: %d, filters: %d\n", numImgsPerGroup, numFiltersPerGroup);
    printf("Image size: %dx%d, filter size: %dx%d\n", imgSize, imgSize, filterSize, filterSize);
    printf("Output grid: %dx%d\n", numOutputsX, numOutputsX);
    printf("Color: %s\n", color ? "yes" : "no");

    int colorMult = color ? 3 : 1;
    Matrix filters(numFiltersPerGroup * numGroups, filterPixels * colorMult);
    Matrix images(numImgsPerGroup * numGroups, imgPixels * colorMult);
    Matrix targets(order == GROUP_FILTER_IMAGE ? numFiltersPerGroup * numGroups : numImgsPerGroup * numGroups,
                   order == GROUP_FILTER_IMAGE ? numImgsPerGroup * numOutputs   : numFiltersPerGroup * numOutputs);
    filters.randomizeUniform();
    images.randomizeUniform();
    targets.apply(Matrix::ZERO);

    NVMatrix nvFilters(filters, true);
    NVMatrix nvImages(images, true);
    NVMatrix nvTargets(targets, true); // eh why not

    //    cutilCheckError(cutResetTimer(timer));
    //    cutilCheckError(cutStartTimer(timer));
    if(color) {
        if (order == GROUP_FILTER_IMAGE) {
            convColorCPU_gfi(images.getData(), filters.getData(), targets.getData(), imgSize, filterSize, numImgsPerGroup, numFiltersPerGroup, numGroups);
        } else {
            convColorCPU_igf(images.getData(), filters.getData(), targets.getData(), imgSize, filterSize, numImgsPerGroup, numFiltersPerGroup, numGroups);
        }
    } else {
        if (order == GROUP_FILTER_IMAGE) {
            convCPU_gfi(images.getData(), filters.getData(), targets.getData(), imgSize, filterSize, numImgsPerGroup, numFiltersPerGroup, numGroups);
        } else {
            convCPU_igf(images.getData(), filters.getData(), targets.getData(), imgSize, filterSize, numImgsPerGroup, numFiltersPerGroup, numGroups);
        }
    }
    //cutilCheckError(cutStopTimer(timer));
    printf("CPU (partial) result:\n");
    targets.print(0, 3, 0, 6);

    printf("CPU is done.\n");
    //printf("CPU time: %.6f msec\n", cutGetTimerValue(timer));

    //    cutilCheckError(cutResetTimer(timer));
    //    cutilCheckError(cutStartTimer(timer));

    convolve(&nvImages, &nvFilters, &nvTargets, numGroups, color, order);

    hipDeviceSynchronize();
    //cutilCheckError(cutStopTimer(timer));
    printf("GPU (partial) result:\n");
    nvTargets.print(0, 3, 0, 6);
    //printf("GPU time: %.6f msec\n", cutGetTimerValue(timer));
    printf("GPU is done.\n");

    // Compare results
    Matrix cpuNVTargets(targets);
    nvTargets.copyToHost(cpuNVTargets);
    cpuNVTargets.subtract(targets);
    cpuNVTargets.apply(Matrix::ABS);
    printf("Max diff between CPU/GPU: %.6f\n", cpuNVTargets.max());
}

#endif //FULL




#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include "cudamat_kernels.cuh"
#include "cudamat.cuh"

extern "C" {

  // just do it, man. It'll be perfectly fine. Don't give up. 
  extern int conv(cudamat* Images, cudamat* Filters, cudamat* Targets, int numGroups, int color, int iorder){    
    ORDER order = iorder==0 ? GROUP_FILTER_IMAGE : IMAGE_GROUP_FILTER;

    //printf("Images ->size[1]=%d, Images ->size[0]=%d\n", Images ->size[1], Images ->size[0]);
    //printf("Filters ->size[1]=%d, Filters ->size[0]=%d\n", Filters ->size[1], Filters ->size[0]);

    NVMatrix nvImages (Images ->data_device, Images ->size[1], Images ->size[0], false);
    NVMatrix nvFilters(Filters->data_device, Filters->size[1], Filters->size[0], false);// is_trans=false
    NVMatrix nvTargets(Targets->data_device, Targets->size[1], Targets->size[0], false);

    //printf("nvImages.getNumCols()=%d, nvImages.getNumRows()=%d\n", nvImages.getNumCols(), nvImages.getNumRows());
    //printf("nvFilters.getNumCols()=%d, nvFilters.getNumRows()=%d\n", nvFilters.getNumCols(), nvFilters.getNumRows());


    convolve(&nvImages, &nvFilters, &nvTargets, numGroups, color, order);

    hipDeviceSynchronize();


    return 0;
  }
  extern int conv2(cudamat* Images, cudamat* Filters, cudamat* Targets, int filterSize, int numGroups, int color, int iorder){    
    ORDER order = iorder==0 ? GROUP_FILTER_IMAGE : IMAGE_GROUP_FILTER;


    NVMatrix nvImages (Images ->data_device, Images ->size[1], Images ->size[0], false);
    NVMatrix nvFilters(Filters->data_device, Filters->size[1], Filters->size[0], false);// is_trans=false
    NVMatrix nvTargets(Targets->data_device, Targets->size[1], Targets->size[0], false);

    //printf("nvImages.getNumCols()=%d, nvImages.getNumRows()=%d\n", nvImages.getNumCols(), nvImages.getNumRows());
    //printf("nvFilters.getNumCols()=%d, nvFilters.getNumRows()=%d\n", nvFilters.getNumCols(), nvFilters.getNumRows());

    convolve2(&nvImages, &nvFilters, &nvTargets, filterSize, numGroups, color, order);

    hipDeviceSynchronize();


    return 0;
  }

  extern int conv3(cudamat* Images, cudamat* Filters, cudamat* Targets, int numGroups, int color, int iorder){    
    ORDER order = iorder==0 ? GROUP_FILTER_IMAGE : IMAGE_GROUP_FILTER;

    NVMatrix nvFilters(Filters->data_device, Filters->size[1], Filters->size[0], false);// is_trans=false
    NVMatrix nvImages(Images->data_device, Images->size[1], Images->size[0], false);
    NVMatrix nvTargets(Targets->data_device, Targets->size[1], Targets->size[0], false);

    convolve3(&nvImages, &nvFilters, &nvTargets, numGroups, color, order);

    hipDeviceSynchronize();


    return 0;
  }

  extern int rot180(cudamat* Filters, cudamat* Targets, int color){
    NVMatrix nvFilters(Filters->data_device, Filters->size[1], Filters->size[0], false);
    NVMatrix nvTargets(Targets->data_device, Targets->size[1], Targets->size[0], false);
    rotate180(&nvFilters, &nvTargets, color);

    hipDeviceSynchronize();

    return 0;
  }

  extern int copy_into_center(cudamat* Images, cudamat* Targets, int paddingSize, int color){
    NVMatrix nvImages( Images->data_device,  Images->size[1],  Images->size[0],  false);
    NVMatrix nvTargets(Targets->data_device, Targets->size[1], Targets->size[0], false);
    copyInto(&nvImages, &nvTargets, paddingSize, color);

    hipDeviceSynchronize();

  }

  extern int add_into_center(cudamat* Images, cudamat* Targets, int paddingSize, int color){
    NVMatrix nvImages( Images->data_device,  Images->size[1],  Images->size[0],  false);
    NVMatrix nvTargets(Targets->data_device, Targets->size[1], Targets->size[0], false);
    addInto(&nvImages, &nvTargets, paddingSize, color);

    hipDeviceSynchronize();

  }

  extern int copy_out_of_center(cudamat* Images, cudamat* Targets, int paddingSize, int color){
    NVMatrix nvImages( Images->data_device,  Images->size[1],  Images->size[0],  false);
    NVMatrix nvTargets(Targets->data_device, Targets->size[1], Targets->size[0], false);
    copyOutOf(&nvImages, &nvTargets, paddingSize, color);

    hipDeviceSynchronize();

  }

  extern int add_out_of_center(cudamat* Images, cudamat* Targets, int paddingSize, int color){
    NVMatrix nvImages( Images->data_device,  Images->size[1],  Images->size[0],  false);
    NVMatrix nvTargets(Targets->data_device, Targets->size[1], Targets->size[0], false);
    addOutOf(&nvImages, &nvTargets, paddingSize, color);

    hipDeviceSynchronize();

  }

  //
  extern int sub_sample(cudamat* Images, cudamat* Targets, int factor){
    NVMatrix nvImages( Images->data_device,  Images->size[1],  Images->size[0],  false);
    NVMatrix nvTargets(Targets->data_device, Targets->size[1], Targets->size[0], false);
    subsample(&nvImages, &nvTargets, factor);

    hipDeviceSynchronize();

  }

  extern int super_sample(cudamat* Images, cudamat* Targets, int factor){
    NVMatrix nvImages( Images->data_device,  Images->size[1],  Images->size[0],  false);
    NVMatrix nvTargets(Targets->data_device, Targets->size[1], Targets->size[0], false);
    supersample(&nvImages, &nvTargets, factor);

    hipDeviceSynchronize();

  }
  //

  extern int matrix_to_grid(cudamat* Images, cudamat* Targets, int squareSize){
    NVMatrix nvImages( Images->data_device,  Images->size[1],  Images->size[0],  false);
    NVMatrix nvTargets(Targets->data_device, Targets->size[1], Targets->size[0], false);
    matrixToGrid(&nvImages, &nvTargets, squareSize, true);

    hipDeviceSynchronize();

  }

  extern int grid_to_matrix(cudamat* Images, cudamat* Targets, int squareSize){
    NVMatrix nvImages( Images->data_device,  Images->size[1],  Images->size[0],  false);
    NVMatrix nvTargets(Targets->data_device, Targets->size[1], Targets->size[0], false);
    gridToMatrix(&nvImages, &nvTargets, squareSize, true);

    hipDeviceSynchronize();

  }



  //#ifdef FULL
extern int run_test_convolve(){
    int boardNum = get_board_lock();
    if (boardNum == GPU_LOCK_NO_BOARD) {
        printf("No free GPU boards!\n");
        exit(EXIT_FAILURE);
    } else if(boardNum == GPU_LOCK_NO_SCRIPT) {
        printf("Running on default board.\n");
    } else {
        printf("Running on board %d\n", boardNum);
    }
    init_tests(boardNum);

    test_convolve(32, 9, true);
    return 0;
}
  //#endif

/* ------------------------------ CUBLAS init/shutdown ------------------------------ */

inline bool check_cublas_error() {
    hipblasStatus_t status = cublasGetError();

    return status != HIPBLAS_STATUS_SUCCESS;
}

inline bool checkCUDAError() {
    hipError_t err = hipGetLastError();

    if (hipSuccess != err)
        printf("%s\n", hipGetErrorString( err));
    return hipSuccess != err;
}

extern const char* get_last_cuda_error() {
    hipError_t err = hipGetLastError();

    return hipGetErrorString( err);
}

extern int cublas_init() {
    cublasInit();
    if (check_cublas_error())
        return CUBLAS_ERROR;
    else
        return 0;
}

extern int cublas_shutdown() {
    cublasShutdown();
    if (check_cublas_error())
        return CUBLAS_ERROR;
    else
        return 0;
}


extern int cuda_set_device(int deviceId) {
    hipSetDevice(deviceId);
    
    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

extern int init_random(rnd_struct* rnd_state, int seed, char* cudamatpath) {
    unsigned int * host_mults;
    host_mults = (unsigned int*)malloc(NUM_RND_STREAMS * sizeof(unsigned int));
    FILE * pFile;

    pFile = fopen (cudamatpath,"r");

    for (int i = 0; i < NUM_RND_STREAMS; i++) {
        fscanf (pFile, "%u", &host_mults[i]);
    }
    fclose (pFile);

    cublasAlloc(NUM_RND_STREAMS, sizeof(unsigned int), (void**)&rnd_state->dev_mults);
    cublasAlloc(NUM_RND_STREAMS, sizeof(unsigned long long), (void**)&rnd_state->dev_words);
    hipblasSetVector(NUM_RND_STREAMS, sizeof(unsigned int), host_mults, 1, rnd_state->dev_mults, 1);
    //hipMalloc((void **)&rnd_state->dev_mults, NUM_RND_STREAMS * sizeof(unsigned int));
    //hipMalloc((void **)&rnd_state->dev_words, NUM_RND_STREAMS * sizeof(unsigned long long));
    //hipMemcpy(rnd_state->dev_mults, host_mults, NUM_RND_STREAMS * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    cudamat_kSeedRandom<<<NUM_RND_BLOCKS, NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, seed);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

/* ------------------------------ Utility routines ------------------------------ */

extern int get_leading_dimension(cudamat* mat) {
    return mat->is_trans ? mat->size[1] : mat->size[0];
}

extern int get_nonleading_dimension(cudamat* mat) {
    return mat->is_trans ? mat->size[0] : mat->size[1];
}

extern void set_transpose(cudamat* mat, int is_trans) {
    mat->is_trans = is_trans;
}

inline char get_transpose_char(cudamat* mat) {
    return mat->is_trans ? 't' : 'n';
}

extern void cuda_sync_threads() {
    hipDeviceSynchronize();
}

/* ------------------------------ Allocating/moving data ------------------------------ */

extern int allocate_device_memory(cudamat* mat) {
    int len = mat->size[0]*mat->size[1];

    hipblasStatus_t stat;

    stat = cublasAlloc(len, sizeof(mat->data_device[0]), (void**)&mat->data_device);

    if (stat != HIPBLAS_STATUS_SUCCESS || check_cublas_error()) {
        checkCUDAError();
        return CUBLAS_ERROR;
    }

    mat->on_device = 1;
    return 0;
}

extern int copy_to_host(cudamat* mat) {
    int len = mat->size[0]*mat->size[1];

    if (mat->on_device) {
            hipblasGetVector(len, sizeof(mat->data_host[0]), mat->data_device, 1, mat->data_host, 1);

        if (check_cublas_error())
            return CUBLAS_ERROR;
    } else
       return ERROR_NOT_ON_DEVICE;
 
    return 0;
}




extern int copy_to_device(cudamat* mat) {
    int len = mat->size[0]*mat->size[1];
    int err_code = 0;

    //if (!mat->owns_data)
    //    return VIEW_ERROR;

    if (!mat->on_device) {
        err_code = allocate_device_memory(mat);
        if (err_code)
            return err_code;
    }

    hipblasSetVector(len, sizeof(mat->data_host[0]), mat->data_host, 1, mat->data_device, 1);
    
    if (check_cublas_error())
        return CUBLAS_ERROR;

    return 0;
}

extern int copy_on_device(cudamat* mat1, cudamat* mat2) {
    int len = mat1->size[0]*mat1->size[1];

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    hipblasScopy(len, mat1->data_device, 1, mat2->data_device, 1);

    if (check_cublas_error())
        return CUBLAS_ERROR;
    else
        return 0;
}

extern int get_row_slice(cudamat* source, cudamat* target, unsigned int start, unsigned int end) {
    int height = source->size[0];
    int width = source->size[1];

    if ((end - start) != target->size[0] || source->size[1] != target->size[1] || start >= end || end > height)
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    dim3 kernelBlockGrid((int)ceil((end - start)/32.), (int)ceil(width/32.), 1);
    dim3 kernelBlockDim(32, 1, 1);

    cudamat_kGetRowSlice<<<kernelBlockGrid,kernelBlockDim>>>(source->data_device, target->data_device, start, end, width, height);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

extern int set_row_slice(cudamat* source, cudamat* target, unsigned int start, unsigned int end) {
    int height = target->size[0];
    int width = target->size[1];

    if ((end - start) != source->size[0] || source->size[1] != target->size[1] || start >= end || end > height)
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    dim3 kernelBlockGrid((int)ceil((end - start)/32.), (int)ceil(width/32.), 1);
    dim3 kernelBlockDim(32, 1, 1);

    cudamat_kSetRowSlice<<<kernelBlockGrid,kernelBlockDim>>>(source->data_device, target->data_device, start, end, width, height);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

extern int copy_transpose(cudamat* source, cudamat* target) {
    unsigned int height = source->size[0];
    unsigned int width = source->size[1];

    if (source->size[0] != target->size[1] || source->size[1] != target->size[0])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    // setup execution parameters
    unsigned int grid_x = height / COPY_BLOCK_SIZE;
    if (height % COPY_BLOCK_SIZE)
        grid_x++;

    unsigned int grid_y = width / COPY_BLOCK_SIZE;
    if (width % COPY_BLOCK_SIZE)
        grid_y++;

    dim3 grid(grid_x, grid_y, 1);
    dim3 threads(COPY_BLOCK_SIZE, COPY_BLOCK_SIZE, 1);

    cudamat_kTranspose<<< grid, threads >>>(target->data_device, source->data_device, height, width);

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

extern int free_device_memory(cudamat* mat) {
    if (mat->owns_data && mat->on_device) {
        hipblasStatus_t stat;

        stat = cublasFree(mat->data_device);
        mat->on_device = 0;

        if (stat != HIPBLAS_STATUS_SUCCESS || check_cublas_error())
            return CUBLAS_ERROR;
    }

    return 0;
}

extern int reshape(cudamat* mat, unsigned int m, unsigned int n) {
    if (mat->size[0] * mat->size[1] != m * n)
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    mat->size[0] = m;
    mat->size[1] = n;

    return 0;
}

extern int get_slice(cudamat* source, cudamat* target, unsigned int first_col, unsigned int last_col) {
    if (source->is_trans)
        return ERROR_TRANSPOSED;

    if (!source->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (last_col > source->size[1] || (first_col >= last_col))
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    int num_rows = source->size[0];

    target->data_host = 0;
    target->data_device = source->data_device + first_col * num_rows;
    target->on_device = 1;
    target->on_host = 0;
    target->size[0] = source->size[0];
    target->size[1] = last_col - first_col;
    target->is_trans = 0;
    target->owns_data = 0;

    return 0;
}

extern int get_vector_slice(cudamat* source, cudamat* target, unsigned int first_ind, unsigned int last_ind) {
    // source must be a vector
    if (source->size[0] > 1 && source->size[1] > 1)
        return ERROR_GENERIC;

    if (source->is_trans)
        return ERROR_TRANSPOSED;

    if (!source->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (first_ind >= last_ind)
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    int num_rows = source->size[0];

    target->data_host = 0;
    target->data_device = source->data_device + first_ind * num_rows;
    target->on_device = 1;
    target->on_host = 0;
    target->is_trans = 0;
    target->owns_data = 0;

    if (source->size[0] > 1) {
        if (last_ind > source->size[0])
            return ERROR_INCOMPATIBLE_DIMENSIONS;

        target->size[0] = last_ind - first_ind;
        target->size[1] = 1;
    } else {
        if (last_ind > source->size[1])
            return ERROR_INCOMPATIBLE_DIMENSIONS;

        target->size[0] = 1;
        target->size[1] = last_ind - first_ind;
    }

    return 0;
}

/* ------------------------------ Initialization routines ------------------------------ */

///// ADDED BY IS FOR THEANO COMPATIBILITY
extern void init_from_cuda_ndarray(cudamat* mat, long gpu_pointer, int m, int n){
  mat->on_host=0;
  mat->size[0]=m;
  mat->size[1]=n;
  mat->on_device=1;
  mat->is_trans=0;
  mat->owns_data=0; 
  mat->data_device = (float*) ((void*) gpu_pointer);
}

extern void init_from_array(cudamat* mat, float* data, int m, int n) {
    mat->data_host = data;
    mat->size[0] = m;
    mat->size[1] = n;
    mat->on_device = 0;
    mat->on_host = 1;
    mat->is_trans = 0;
    mat->owns_data = 1;
}

extern int init_empty(cudamat* mat, int m, int n) {
    mat->size[0] = m;
    mat->size[1] = n;
    mat->on_device = 0;
    mat->on_host = 0;
    mat->is_trans = 0;
    mat->owns_data = 1;

    return allocate_device_memory(mat);
}

/* ------------------------------ Random number generation ------------------------------ */
extern int fill_with_rand(rnd_struct* rnd_state, cudamat* mat) {
    int len = mat->size[0] * mat->size[1];

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    cudamat_kRandomUniform<<<NUM_RND_BLOCKS,NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, mat->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}

extern int fill_with_randn(rnd_struct* rnd_state, cudamat* mat) {
    int len = mat->size[0] * mat->size[1];

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    cudamat_kRandomGaussian<<<NUM_RND_BLOCKS,NUM_RND_THREADS_PER_BLOCK>>>(rnd_state->dev_mults, rnd_state->dev_words, mat->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}
/* ------------------------------ Algebraic operations ------------------------------ */

extern int add_col_vec(cudamat* mat, cudamat* vec, cudamat* target) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[0] != vec->size[0] || vec->size[1] != 1 ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kAddColVector<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, w, h);

    hipDeviceSynchronize();

    if (checkCUDAError()) {
        return CUDA_ERROR;
    }

    return 0;
}

extern int add_col_mult(cudamat* mat, cudamat* vec, cudamat* target, float mult) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[0] != vec->size[0] || vec->size[1] != 1 ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kAddColMult<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, mult, w, h);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int add_row_vec(cudamat* mat, cudamat* vec, cudamat* target) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[1] != vec->size[1] || vec->size[0] != 1 ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kAddRowVector<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, w, h);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int mult_by_col_vec(cudamat* mat, cudamat* vec, cudamat* target) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[0] != vec->size[0] || vec->size[1] != 1 ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kMultByColVector<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, w, h);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int mult_by_row_vec(cudamat* mat, cudamat* vec, cudamat* target) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !vec->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (mat->size[1] != vec->size[1] || vec->size[0] != 1 ||
        mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kMultByRowVector<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, vec->data_device, target->data_device, w, h);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int less_than(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kLessThan<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int less_than_scalar(cudamat* mat, float val, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans != target->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kLessThanScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, val, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int greater_than(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kGreaterThan<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int greater_than_scalar(cudamat* mat, float val, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans != target->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kGreaterThanScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, val, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int max_by_axis(cudamat* mat, cudamat* target, int axis) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (axis == 0) {
        if (target->size[0] != 1 || target->size[1] != mat->size[1])
            return ERROR_INCOMPATIBLE_DIMENSIONS;

        cudamat_kMaxColumnwise<<<w,32>>>(mat->data_device, target->data_device, w, h);

        hipDeviceSynchronize();
    } else
        return ERROR_UNSUPPORTED;

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}


extern int max_row_argmax(cudamat* mat, cudamat* target_max, cudamat* target_argmax, int axis) {
    unsigned int h = mat->size[0],
                 w = mat->size[1];

    if (!mat->on_device || !target_max->on_device || !target_argmax->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans)
        return ERROR_TRANSPOSED;

    if (axis == 0) {
        if (target_max->size[0] != 1 || 
	    target_max->size[1] != mat->size[1] || 
	    target_max->size[1] != target_argmax->size[1] ||
	    mat->size[0] != target_argmax->size[0])
            return ERROR_INCOMPATIBLE_DIMENSIONS;

        cudamat_kRowArgmax<<<w,32>>>(mat->data_device, 
				     target_max->data_device, 
				     target_argmax->data_device, 
				     w, h);

        hipDeviceSynchronize();
    } else
        return ERROR_UNSUPPORTED;

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}



extern int sign(cudamat* mat, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->is_trans != target->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kSign<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_sigmoid(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kApplySigmoid<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_tanh(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kApplyTanh<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_abs(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kApplyAbs<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_log_1_plus_exp(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kApplyLog1PlusExp<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_log(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kLog<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_exp(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kExp<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_sqrt(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kSqrt<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_pow(cudamat* mat, float pow, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kPow<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, pow, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int apply_pow_matrix(cudamat* mat, cudamat* pow, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    if (mat->size[0] != pow->size[0] || mat->size[1] != pow->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kPowMatrix<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, pow->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int reciprocal(cudamat* mat, cudamat* target) {
    unsigned int len = mat->size[0] * mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kReciprocal<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int dot(cudamat* mat1, cudamat* mat2, cudamat* target, float beta, float alpha) {
    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (get_leading_dimension(mat1) != get_leading_dimension(target) ||
        get_nonleading_dimension(mat2) != get_nonleading_dimension(target) ||
        get_nonleading_dimension(mat1) != get_leading_dimension(mat2)) {
        return ERROR_INCOMPATIBLE_DIMENSIONS;
    }
    int m = get_leading_dimension(mat1),
        k = get_leading_dimension(mat2),
        n = get_nonleading_dimension(mat2);

    hipblasSgemm(get_transpose_char(mat1), get_transpose_char(mat2), 
                m, n, k,
                alpha, mat1->data_device, mat1->size[0],
                mat2->data_device, mat2->size[0],
                beta, target->data_device, target->size[0]);

    hipDeviceSynchronize();

    if (check_cublas_error())
        return CUBLAS_ERROR;

    return 0;
}

extern float vdot(cudamat* mat1, cudamat* mat2, int* err_code) {
    int len = mat1->size[0]*mat1->size[1];
    float res;

    if (!mat1->on_device || !mat2->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans) {
        *err_code = ERROR_TRANSPOSEDNESS;
        return 0;
    }

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1]) { 
        *err_code = ERROR_INCOMPATIBLE_DIMENSIONS;
        return 0;
    }

    res = hipblasSdot(len, mat1->data_device, 1, mat2->data_device, 1);

    if (check_cublas_error()) {
        *err_code = CUBLAS_ERROR;
        return -1.;
    } else {
        *err_code = 0;
        return res;
    }
}

/* Perform the operation mat1 = mat1 + alpha * mat2. mat1 and mat2 must
   have the same transposedness. */
extern int add_mult(cudamat* mat1, cudamat* mat2, float alpha) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    hipblasSaxpy(len, alpha, mat2->data_device, 1, mat1->data_device, 1);

    if (check_cublas_error())
        return CUBLAS_ERROR;

    return 0;
}

extern int add_elementwise(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kAdd<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int subtract_elementwise(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kSubtract<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int divide_elementwise(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kDivide<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

/* Elementwise multiplication of 2 matrices */
extern int mult_elementwise(cudamat* mat1, cudamat* mat2, cudamat* target) {
    int len = mat1->size[0]*mat1->size[1];

    if (!mat1->on_device || !mat2->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat1->is_trans != mat2->is_trans)
        return ERROR_TRANSPOSEDNESS;

    if (mat1->size[0] != mat2->size[0] || mat1->size[1] != mat2->size[1] ||
        mat1->size[0] != target->size[0] || mat1->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kMult<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat1->data_device, mat2->data_device, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int assign_scalar(cudamat* mat, float alpha) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    cudamat_kAssignScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, alpha, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int mult_by_scalar(cudamat* mat, float alpha, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kMultScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, alpha, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int divide_by_scalar(cudamat* mat, float alpha, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kDivideScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, alpha, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern int add_scalar(cudamat* mat, float alpha, cudamat* target) {
    int len = mat->size[0]*mat->size[1];

    if (!mat->on_device || !target->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (mat->size[0] != target->size[0] || mat->size[1] != target->size[1])
        return ERROR_INCOMPATIBLE_DIMENSIONS;

    cudamat_kAddScalar<<<NUM_VECTOR_OP_BLOCKS,NUM_VECTOR_OP_THREADS_PER_BLOCK>>>(mat->data_device, alpha, target->data_device, len);

    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;

    return 0;
}

extern float euclid_norm(cudamat* mat, int* err_code) {
    int len = mat->size[0]*mat->size[1];

    float res =  hipblasSnrm2(len, mat->data_device, 1);

    if (!mat->on_device)
        return ERROR_NOT_ON_DEVICE;

    if (check_cublas_error()) {
        *err_code = CUBLAS_ERROR;
        return -1.;
    } else {
        *err_code = 0;
        return res;
    }
}

// extern int selectRows(cudamat* source, cudamat* target, cudamat* indices){
//     const int nRetRows = indices->size[1];

//     if (nRetRows==0) return 0;

//     dim3 gridDim((nRetRows+31)/32);
//     dim3 blockDim(32);

//     cudamat_kSelectRows<<<gridDim, blockDim>>>(source->data_device, target->data_device, indices->data_device, nRetRows, source->size[0], source->size[1]);
//     hipDeviceSynchronize();

//     if (checkCUDAError())
//         return CUDA_ERROR;
//     else
//         return 0;
// }

extern int setSelectedRows(cudamat* target, cudamat* source, cudamat* indices){
    const int nSetRows = indices->size[1];

    if (nSetRows==0)
        return 0;

    dim3 gridDim((nSetRows+31)/32);
    dim3 blockDim(32);

    cudamat_kSetSelectedRows<<<gridDim, blockDim>>>(target->data_device, source->data_device, indices->data_device, nSetRows, target->size[0], target->size[1]);
    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}


extern int clfVsProduct(int nComponents, int vectorLength, int nothingIndex_scalars, cudamat* inVectors, cudamat* outVectors, cudamat* globalScalars, cudamat* inIndices, int nOutputs){
  int gridX=nOutputs, gridY=1;
  while (gridX>65535) {gridY*=2; gridX = (gridX+1)/2;}
  dim3 gridDim(gridX, gridY);
  if (nOutputs>0) cudamat_kClfVsProduct<<<gridDim, max(32, vectorLength)>>>(nComponents, vectorLength, nothingIndex_scalars,
								    inVectors->data_device, outVectors->data_device, globalScalars->data_device, inIndices->data_device, nOutputs);
  hipDeviceSynchronize();
  if (checkCUDAError()) return CUDA_ERROR; else return 0;
}


extern int clfPcOuterProduct(cudamat* indexPairs, cudamat* nIndexPairss, cudamat* A, cudamat* B, cudamat* ret){
  const int nCols = ret->size[0];
  const int nRows = ret->size[1];
  int gridX=nRows, gridY=1;
  while (gridX>65535) {gridY*=2; gridX = (gridX+1)/2;}
  dim3 gridDim(gridX, gridY);
  const int Ns = indexPairs->size[0]*4;
  cudamat_kClfPcOuterProduct<<<gridDim, max(32, nCols), Ns>>>(indexPairs->size[0]/2, indexPairs->data_device, nIndexPairss->data_device, A->data_device, B->data_device, ret->data_device, nCols, nRows);
  hipDeviceSynchronize();
  if (checkCUDAError()) return CUDA_ERROR; else return 0;
}


extern int selectRows(cudamat* source, cudamat* target, cudamat* indices){
    const int nRetRows = indices->size[1];

    if (nRetRows==0) return 0;

    int gridX=(nRetRows+31)/32, gridY=1;
    while (gridX>65535) {gridY*=2; gridX = (gridX+1)/2;}
    dim3 gridDim(gridX, gridY);

    dim3 blockDim(32);

    cudamat_kSelectRows<<<gridDim, blockDim>>>(source->data_device, target->data_device, indices->data_device, nRetRows, source->size[0], source->size[1]);
    hipDeviceSynchronize();

    if (checkCUDAError())
        return CUDA_ERROR;
    else
        return 0;
}




}









